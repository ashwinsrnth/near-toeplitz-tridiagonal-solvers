
#include <hip/hip_runtime.h>
__global__ void sharedMemCyclicReduction( double *a_g,
                                double *b_g,
                                double *c_g,
                                double *d_g,
                               int nx,
                               int ny,
                               int nz,
                               int bx,
                               int by) {
    /*
        Solve several systems by cyclic reduction,
        each of size block_size.
        
        bx and by are the block size.
        Specifically, they do not refer to the actual
        number of threads per block.
        bx = nx
        but nx/2 threads per block are launched.
    */
    __shared__ double a_l[1024];
    __shared__ double b_l[1024];
    __shared__ double c_l[1024];
    __shared__ double d_l[1024];
    int ix = blockIdx.x*blockDim.x + threadIdx.x; 
    int iy = blockIdx.y*blockDim.y + threadIdx.y; 
    int iz = blockIdx.z*blockDim.z + threadIdx.z; 
    int lix = threadIdx.x; 
    int liy = threadIdx.y; 
    int liz = threadIdx.z; 
    int i, m, n;
    int stride;

    int i3d = iz*(nx*ny) + iy*nx + ix;
    int i3d0 = iz*(nx*ny) + iy*nx + 0;
    int li3d = liz*(bx*by) + liy*bx + lix;
    int li3d0 = liz*(bx*by) + liy*bx + 0;

    double k1, k2;
    double d_m, d_n;

    /* each block reads two elements to shared memory */
    a_l[li3d0+2*lix]  = a_g[2*lix];
    a_l[li3d0+2*lix+1] = a_g[2*lix+1];
    b_l[li3d0+2*lix]  = b_g[2*lix];
    b_l[li3d0+2*lix+1] = b_g[2*lix+1];
    c_l[li3d0+2*lix]  = c_g[2*lix];
    c_l[li3d0+2*lix+1] = c_g[2*lix+1];
    d_l[li3d0+2*lix] = d_g[i3d0+2*lix];
    d_l[li3d0+2*lix+1] = d_g[i3d0+2*lix+1];
    __syncthreads();
    
    /* solve the block in shared memory */
    stride = 1;
    for (int step=0; step<rint(log2((float) nx)); step++) {
        stride = stride*2;

        if (lix < nx/stride) {
            i = (stride-1) + lix*stride;
            ix = li3d0 + i;

            if (stride == nx) {
                m = li3d0 + nx/2 - 1;
                n = li3d0 + nx - 1;

                d_m = (d_l[m]*b_l[n] - c_l[m]*d_l[n])/(b_l[m]*b_l[n] - c_l[m]*a_l[n]);
                d_n = (b_l[m]*d_l[n] - d_l[m]*a_l[n])/(b_l[m]*b_l[n] - c_l[m]*a_l[n]);
                d_l[m] = d_m;
                d_l[n] = d_n;
            }

            else {
                if (i == (nx-1)) {
                    ix = li3d0 + i;
                    k1 = a_l[ix]/b_l[ix-stride/2];
                    a_l[ix] = -a_l[ix-stride/2]*k1;
                    b_l[ix] = b_l[ix] - c_l[ix-stride/2]*k1;
                    d_l[ix] = d_l[ix] - d_l[ix-stride/2]*k1;
                }
                else {
                    k1 = a_l[ix]/b_l[ix-stride/2];
                    k2 = c_l[ix]/b_l[ix+stride/2];
                    a_l[ix] = -a_l[ix-stride/2]*k1;
                    b_l[ix] = b_l[ix] - c_l[ix-stride/2]*k1 - a_l[ix+stride/2]*k2;
                    c_l[ix] = -c_l[ix+stride/2]*k2;
                    d_l[ix] = d_l[ix] - d_l[ix-stride/2]*k1 - d_l[ix+stride/2]*k2;
                }
            }
        }
        __syncthreads();
    }


    
    for (int step=0; step<rint(log2((float) nx))-1; step++) {
        stride = stride/2;

        if (lix < nx/stride){
            i = (stride/2-1) + lix*stride;
            ix = li3d0 + i;

            if (i < stride) {
                d_l[ix] = (d_l[ix] - c_l[ix]*d_l[ix+stride/2])/b_l[ix];
            }

            else {
                d_l[ix] = (d_l[ix] - a_l[ix]*d_l[ix-stride/2] - c_l[ix]*d_l[ix+stride/2])/b_l[ix];
            }
        }

        __syncthreads();
    }
    
    /* write from shared memory to x_d */
    
    d_g[i3d0+2*lix] = d_l[li3d0+2*lix];
    d_g[i3d0+2*lix+1] = d_l[li3d0+2*lix+1];
    __syncthreads();
}

